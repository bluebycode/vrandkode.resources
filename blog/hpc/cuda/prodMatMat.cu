#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void
vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int start_time = clock();
    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
    unsigned int stop_time = clock();

    if(i == 0)
    {
    printf("Time spent: %d\n", stop_time - start_time);

    }
}

__global__ void
matrixAdd(const float *A, const float *B, float *C, int size)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int start_time = clock();
    if (idx < size)
    {
        for (int i = 0; i < size; i++)
        {
            C[idx*size + i] =  A[idx*size + i] + B[idx*size + i];
        }
    }
    unsigned int stop_time = clock();

    if (idx == 0)
    {
       printf("Time spent: %d\n", stop_time - start_time);
    }
}

__global__ void
prodMV(const float *M, const float *V, float *C, int size)
{
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    unsigned int start_time = clock();

    float sum = 0.0f;
    if (row < size && col < size)
    {
        for (int i = 0; i < vSize; i++)
        {
          sum += A[row * size + i] * B[i * size + col];
        }
    }
    C[row * size + col] = sum;

    unsigned int stop_time = clock();

    if(row == 0)
    {
       printf("Time spent: %d\n", stop_time - start_time);
    }
}

/**
 * Host main routine
 */
int
main(void)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = 5000;
    size_t size = numElements * sizeof(float);
    printf("[Matrix addition of %d elements]\n", numElements);

    // Allocate the host input vector A
    float *h_A = (float *)malloc(size*size);

    // Allocate the host input vector B
    float *h_B = (float *)malloc(size*size);

    // Allocate the host output vector C
    float *h_C = (float *)malloc(size*size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // Allocate the device input matrix A
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, size*size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, size*size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, size*size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //
    // vectors -> device input

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size*size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size*size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //
    // Launch the CUDA Kernel

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock; //196
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    matrixAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size*size, hipMemcpyDeviceToHost);


    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    float sum;
    for (int row=0; row<size row++){
        for (int col=0; col<size; col++){
            sum = 0.f;
            for (int n=0; n<size; n++){
                sum += h_A[row * size + n] * h_B[n*size+col];
            }
            cpu_C[row * size + col] = sum;
        }
    }

    for (int row=0; row < size; row++){
        for (int col=0; col < size; col++){
            if (fabs(cpu_C[row * size + col] - h_C[row * size + col]) > 1e-2)
            {
                fprintf(stderr, "Result verification failed at element %d: %1.5f, %1.5f!\n", i, goldC[i], h_C[i]);
                exit(EXIT_FAILURE);
            }
        }
    }

    printf("Test PASSED\n");


    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    // Reset the device and exit
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;
}

