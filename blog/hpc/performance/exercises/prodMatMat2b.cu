/**
 * Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */

__global__ void
prodMul(const float *A, const float *B, float *C, int size)
{
    
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int i = blockDim.y * blockIdx.y + threadIdx.y;

    unsigned int start_time = clock();

    float sum = 0.0f;
    if (i < size && j < size)
    {
        for (int k = 0; k < size; k++)
        {
           sum += A[i * size + k] * B[k*size + j];
        }
        C[i * size + j] = sum;
        //printf("(%d,%d):  %f \n", i, j, C[i*size + j]);
    }

    unsigned int stop_time = clock();

    if(i == 0 && j==0)
    {
       printf("Time spent: %d\n", stop_time - start_time);
    }
}

#define NUM_ELEMENTS 40

/**
 * Host main routine
 */
int
main(void)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = NUM_ELEMENTS*NUM_ELEMENTS;
    size_t size = numElements * sizeof(float);
    printf("[Matrix addition of %d elements]\n", numElements);

    // Allocate the host input vector A
    float *h_A = (float *)malloc(size);

    // Allocate the host input vector B
    float *h_B = (float *)malloc(size);

    // Allocate the host output vector C
    float *h_C = (float *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // Allocate the device input matrix A
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //
    // vectors -> device input

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //
    // Launch the CUDA Kernel

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlockAxis = 16;
    int blocksPerGrid =(NUM_ELEMENTS + threadsPerBlockAxis - 1) / threadsPerBlockAxis; //196
    printf("CUDA kernel launch with %d blocks of (%dx%d) threads\n", blocksPerGrid, threadsPerBlockAxis, threadsPerBlockAxis);
    printf("Total size: %d (N=%d) Total threads: %d (T=%d,B=%d)",numElements, NUM_ELEMENTS, threadsPerBlockAxis*threadsPerBlockAxis*blocksPerGrid*blocksPerGrid, threadsPerBlockAxis, blocksPerGrid);
    dim3 dimBlock(threadsPerBlockAxis, threadsPerBlockAxis, 1);
    dim3 dimGrid(blocksPerGrid, blocksPerGrid, 1);
    prodMul<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, NUM_ELEMENTS);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);


    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    
    float * cpu_C = (float *) malloc (size);
    float sum;
    int N = NUM_ELEMENTS;
    for (int i=0; i<N; i++){
        for (int j=0; j <N; j++){
            sum = 0.f;
            for (int k=0; k<N; k++){
                sum += h_A[i*N + k] * h_B[k*N + j];
            }
            cpu_C[i*N + j] = sum;
            //ºfprintf(stdout, "(%d,%d):%f ? %f", i, j, cpu_C[i*N + j],h_C[i*N + j]);
        }
    }
    unsigned int i = 0;
    for (int col=0; col <N; col++){
        for (int row=0; row <N; row++){
            i = col * N + row;
            if (fabs(cpu_C[i] - h_C[i]) > 1e-2)
            {
                fprintf(stderr, "Result verification failed at element %d (%d,%d): %1.5f, %1.5f!\n", i, row, col, cpu_C[i], h_C[i]);
                exit(EXIT_FAILURE);
            }
        }
    }

    printf("Test PASSED\n");


    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    // Reset the device and exit
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;
}

