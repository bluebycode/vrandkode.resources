/**
 * Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */

__global__ void
prodMul(const float *A, const float *B, float *C, int size)
{
    
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    unsigned int start_time = clock();

    float sum = 0.0f;
    if (i < size)
    {
        for (int j = 0; j < size; j++)
        {
          sum = 0.0f;
          for (int k = 0; k < size; k++)
          {
             sum += A[i * size + k] * B[k*size + j];
          }
          C[i * size + j] = sum;
          //printf("(%d,%d):%f\n", i, j, C[i*size + j]);
        }
    }

    unsigned int stop_time = clock();

    if(i == 0)
    {
       printf("Time spent: %d\n", stop_time - start_time);
    }
}

#define NUM_ELEMENTS 40

/**
 * Host main routine
 */
int
main(void)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = NUM_ELEMENTS;
    size_t size = numElements * sizeof(float);
    printf("[Matrix addition of %d elements]\n", numElements);

    // Allocate the host input vector A
    float *h_A = (float *)malloc(size*size);

    // Allocate the host input vector B
    float *h_B = (float *)malloc(size*size);

    // Allocate the host output vector C
    float *h_C = (float *)malloc(size*size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements*numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // Allocate the device input matrix A
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, size*size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, size*size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, size*size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //
    // vectors -> device input

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size*size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed ºto copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size*size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //
    // Launch the CUDA Kernel

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock; //196
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    prodMul<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size*size, hipMemcpyDeviceToHost);


    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    
    float * cpu_C = (float *) malloc (size*size);
    float sum;
    for (int i=0; i<numElements; i++){
        for (int j=0; j <numElements; j++){
            sum = 0.f;
            for (int k=0; k<numElements; k++){
                sum += h_A[i*numElements + k] * h_B[k*numElements + j];
            }
            cpu_C[i*numElements + j] = sum;
//            fprintf(stdout, "(%d,%d):%f ? %f\n ", i, j, cpu_C[i*numElements + j], h_C[i*numElements + j]);
        }
    }
    unsigned int i = 0;
    for (int col=0; col <numElements; col++){
        for (int row=0; row <numElements; row++){
            i = col * numElements + row;
            if (fabs(cpu_C[i] - h_C[i]) > 1e-2)
            {
                fprintf(stderr, "Result verification failed at element %d (%d,%d): %1.5f, %1.5f!\n", i, row, col, cpu_C[i], h_C[i]);
                exit(EXIT_FAILURE);
            }
        }
    }

    printf("Test PASSED\n");


    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    // Reset the device and exit
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;
}

